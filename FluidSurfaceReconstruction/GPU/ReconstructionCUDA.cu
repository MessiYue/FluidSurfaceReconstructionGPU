#include "hip/hip_runtime.h"

#include <Windows.h>
#include "ReconstructionCUDA.cuh"

#include <hip/hip_runtime.h>
#include <>

#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <thrust/for_each.h>
#include <thrust/device_ptr.h>
#include <hip/hip_cooperative_groups.h>
#include <thrust/iterator/zip_iterator.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <helper_functions.h>

#include "Defines.h"
#include "CudaUtils.h"

using namespace cooperative_groups;

__constant__ float EPSILON_ = (float)1.0e-7;

//! [���8���������ֵ��λ�ù�ϵ��8λ��������]��ֵΪ���12�������ֵ���ཻ����������ֻ�õ�12λ������СΪ256
texture <uint, 1, hipReadModeElementType> edgeTex;
//! [���8���������ֵ��λ�ù�ϵ��8λ��������][���15�����㣨���������Σ���
//! ���һ��Ԫ�����ڽ������]��ֵΪ�����������ζ�������cube�ıߺ�
//! �������������������˳��Ϊ��ʱ�뷽�� ��СΪ256
texture <int, 1, hipReadModeElementType> edgeIndexesOfTriangleTex;
//! ��Ӧcell/voxel�����Ķ���������СΪ256
texture <uint, 1, hipReadModeElementType> numVerticesTex;
//! [�߱��][��������]��ֵΪ�ñߵ����������ţ���СΪ12
texture <uint, 1, hipReadModeElementType> vertexIndexesOfEdgeTex;

//! func: get global thread id.
inline __device__  
uint getThreadIdGlobal()
{
	uint blockId = blockIdx.z*gridDim.y*gridDim.x + blockIdx.y*gridDim.x + blockIdx.x;
	uint threadId = threadIdx.z*blockDim.y*blockDim.x + threadIdx.y*blockDim.x + threadIdx.x 
		+ blockId*blockDim.x*blockDim.y*blockDim.z;
	return threadId;
}

//! func: bind auxiliary textures for marching cubes.
extern "C"
void bindTextures(uint* d_edgeTable, int* d_edgeIndicesOfTriangleTable, 
	uint* d_numVerticesTable, uint* d_vertexIndicesOfEdgeTable)
{
	// texture's channel format.
	hipChannelFormatDesc channelDescUnsigned = 
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
	hipChannelFormatDesc channelDescSigned =
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);

	// data transfer and textures binding.
	checkCudaErrors(hipBindTexture(0, edgeTex, d_edgeTable, channelDescUnsigned));
	checkCudaErrors(hipBindTexture(0, edgeIndexesOfTriangleTex, d_edgeIndicesOfTriangleTable, channelDescSigned));
	checkCudaErrors(hipBindTexture(0, numVerticesTex, d_numVerticesTable, channelDescUnsigned));
	checkCudaErrors(hipBindTexture(0, vertexIndexesOfEdgeTex, d_vertexIndicesOfEdgeTable, channelDescUnsigned));
}

//! --------------------------------------cuda kernel functions------------------------------------------------

//! func: surface vertices' estimation using simple virutal density field.
__global__ 
void estimateSurfaceVertices(
	DensityGrid densityGrid,		// input, virtual density field.
	IsSurfaceGrid isSurfaceGrid,	// output, whether the corresponding grid point is in surface region or not. 
	uint scSearchExt,				// input, search expansion extent.
	SimParam params)				
{
	// get corresponding 3D index.
	uint threadId = getThreadIdGlobal();
	uint3 densityGridRes = densityGrid.resolution;
	uint3 dR_1 = densityGridRes - 1;
	uint3 curIndex3D = index1DTo3D(threadId, densityGridRes);

	// boundary detection.
	if (curIndex3D.x >= dR_1.x || curIndex3D.y >= dR_1.y || curIndex3D.z >= dR_1.z)
		return;

	// get corresponding situation flag.
	uint vertexFlag = getVertexFlag(curIndex3D, densityGrid, 0.5f);

	// ��ǰcube���ֵ���н���
	if (vertexFlag > 0 && vertexFlag < 255)
	{
		int3 minIndex3D = make_int3(curIndex3D.x, curIndex3D.y, curIndex3D.z);
		int3 maxIndex3D = make_int3(curIndex3D.x + 1, curIndex3D.y + 1, curIndex3D.z + 1);

		// expanding.
		minIndex3D = minIndex3D * params.scSpGridResRatio - scSearchExt;
		maxIndex3D = maxIndex3D * params.scSpGridResRatio + scSearchExt;
		
		// clamping.
		minIndex3D = clamp(minIndex3D, make_int3(0, 0, 0), 
			make_int3(isSurfaceGrid.resolution.x - 1, isSurfaceGrid.resolution.y - 1, isSurfaceGrid.resolution.z - 1));
		maxIndex3D = clamp(maxIndex3D, make_int3(0, 0, 0), 
			make_int3(isSurfaceGrid.resolution.x - 1, isSurfaceGrid.resolution.y - 1, isSurfaceGrid.resolution.z - 1));

		// mark corresponding cell as surface cell (let it equals 1).
		for (uint zSc = minIndex3D.z; zSc <= maxIndex3D.z; zSc++)
		{
			for (uint ySc = minIndex3D.y; ySc <= maxIndex3D.y; ySc++)
			{
				for (uint xSc = minIndex3D.x; xSc <= maxIndex3D.x; xSc++)
				{
					uint3 curIndex3DInScalarGrid = make_uint3(xSc, ySc, zSc);
					isSurfaceGrid.grid[index3DTo1D(curIndex3DInScalarGrid, isSurfaceGrid.resolution)] = 1;
				}
			}
		}
	}
}

//! func: calculate the corresponding vertex's scalar value of scalar field using TMC01 method.
__device__ 
void updateScalarFieldValueTC01(
	uint vertexIndex1D,								// input, vertex's index of scalar field grid.
	ParticleIndexRangeGrid particleIndexRangeGrid,	// input, particles' indices for each cell of spatial grid.
	ParticleArray particleArray,					// input, particle array.
	ScalarFieldGrid ScalarFieldGrid,							// output, scalar field grid.
	GridInfo spatialGridInfo,						// input, spatial hasing grid information.
	GridInfo scalarGridInfo,						// input, scalar field grid information.
	SimParam params)					 
{
	// get corresponding vertex position.
	float3 vPos = getVertexPos(index1DTo3D(vertexIndex1D, ScalarFieldGrid.resolution),
		scalarGridInfo.minPos, scalarGridInfo.cellSize);
	int3 curIndex = getIndex3D(vPos, spatialGridInfo.minPos, spatialGridInfo.cellSize);

	// get influenced spatial hashing cells' bounding box and clamping.
	int3 minIndex = curIndex - 1;
	int3 maxIndex = curIndex + 1;
	minIndex = clamp(minIndex, make_int3(0, 0, 0), make_int3(particleIndexRangeGrid.resolution.x - 1,
		particleIndexRangeGrid.resolution.y - 1, particleIndexRangeGrid.resolution.z - 1));
	maxIndex = clamp(maxIndex, make_int3(0, 0, 0), make_int3(particleIndexRangeGrid.resolution.x - 1,
		particleIndexRangeGrid.resolution.y - 1, particleIndexRangeGrid.resolution.z - 1));

	//ScalarValue* v = &ScalarFieldGrid.grid[vertexIndex1D];
	float val = 0.f;
	for (int zSp = minIndex.z; zSp <= maxIndex.z; zSp++)
	{
		for (int ySp = minIndex.y; ySp <= maxIndex.y; ySp++)
		{
			for (int xSp = minIndex.x; xSp <= maxIndex.x; xSp++)
			{
				// 3D index of spatialGrid.
				uint3 index3D = make_uint3(xSp, ySp, zSp);
				// ������particleArray�е�������Ϣ��ƫ���볤�ȣ�
				IndexRange IndexRange = particleIndexRangeGrid.grid[index3DTo1D(index3D, particleIndexRangeGrid.resolution)];
				// travel each particle of the corresponding cell to calcular scalr value.
				if (IndexRange.start == 0xffffffff)
					continue;
				for (uint i = IndexRange.start; i < IndexRange.end; i++)
				{
					float3 delta = vPos - particleArray.grid[i].pos;
					float distSq = dot(delta, delta);
					// using TC01 kernel function.
					val += kernelTC01(distSq, params.smoothingRadiusSq);
				}
			}
		}
	}
	ScalarFieldGrid.grid[vertexIndex1D].value = val;
}

//! func: calculate the corresponding vertex's scalar value of scalar field using ZB05 method.
__device__
void updateScalarFieldValueZB05(
	uint vertexIndex1D,								// input, vertex's index of scalar field grid.
	ParticleIndexRangeGrid particleIndexRangeGrid,	// input, particles' indices for each cell of spatial grid.
	ParticleArray particleArray,					// input, particle array.
	ScalarFieldGrid ScalarFieldGrid,							// output, scalar field grid.
	GridInfo spatialGridInfo,						// input, spatial hasing grid information.
	GridInfo scalarGridInfo,						// input, scalar field grid information.
	SimParam params)
{
	// get corresponding vertex position.
	float3 vPos = getVertexPos(index1DTo3D(vertexIndex1D, ScalarFieldGrid.resolution),
		scalarGridInfo.minPos, scalarGridInfo.cellSize);

	// get influenced spatial hashing cells' bounding box and clamping.
	int3 curIndex = getIndex3D(vPos, spatialGridInfo.minPos, spatialGridInfo.cellSize);
	int3 minIndex = curIndex - 1;
	int3 maxIndex = curIndex + 1;
	minIndex = clamp(minIndex, make_int3(0, 0, 0), make_int3(particleIndexRangeGrid.resolution.x - 1,
		particleIndexRangeGrid.resolution.y - 1, particleIndexRangeGrid.resolution.z - 1));
	maxIndex = clamp(maxIndex, make_int3(0, 0, 0), make_int3(particleIndexRangeGrid.resolution.x - 1,
		particleIndexRangeGrid.resolution.y - 1, particleIndexRangeGrid.resolution.z - 1));

	float wSum = 0.0f;
	float3 posAvg = make_float3(0.0f, 0.0f, 0.0f);
	for (int zSp = minIndex.z; zSp <= maxIndex.z; zSp++)
	{
		for (int ySp = minIndex.y; ySp <= maxIndex.y; ySp++)
		{
			for (int xSp = minIndex.x; xSp <= maxIndex.x; xSp++)
			{
				// 3D index of spatialGrid.
				uint3 index3D = make_uint3(xSp, ySp, zSp);
				IndexRange IndexRange = particleIndexRangeGrid.grid[index3DTo1D(index3D, particleIndexRangeGrid.resolution)];
				// travel each particle of the corresponding cell to calcular scalr value.
				if (IndexRange.start == 0xffffffff)
					continue;
				for (uint i = IndexRange.start; i < IndexRange.end; i++)
				{
					float3 neighborPos = particleArray.grid[i].pos;
					float3 delta = vPos - neighborPos;
					float distSq = dot(delta, delta);

					// using ZB05 kernel function.
					const float wi = kernelZB05(distSq, params.smoothingRadiusSq);
					wSum += wi;
					posAvg += neighborPos * wi;
				}
			}
		}
	}
	if (wSum > 0.0f)
	{
		posAvg /= wSum;
		ScalarFieldGrid.grid[vertexIndex1D].value = length(vPos - posAvg) - params.particleRadius;
	}
}

//! func: call function updateVertexValue() to calculate scalar field just for surface cell.
//! Actually it's not a good idea since all the branch would be executed despite it's surface cell or not.
__global__ 
void updateScalarGridValuesStd(
	IsSurfaceGrid isSurfaceGrid,				// input, whether the corresponding grid point is in surface region or not.
	ParticleIndexRangeGrid particleIndexRangeGrid,// input, particles' indices for each cell of spatial grid.	
	ParticleArray particleArray,				// input, particles array.
	ScalarFieldGrid ScalarFieldGrid,						// output, scalar field grid.
	GridInfo spatialGridInfo,					// input, spatial hasing grid information.
	GridInfo scalarGridInfo,					// input, scalar field grid information.
	SimParam params
	)					
{
	uint threadId = getThreadIdGlobal();
	// boundary detection.
	if (threadId >= ScalarFieldGrid.size)
		return;
	// if the grid point is not in surface region, just return.
	if (isSurfaceGrid.grid[threadId] != 1)
		return;
	// call function updateVertexValue() to calculate scalar field value.
	updateScalarFieldValueTC01(threadId, particleIndexRangeGrid, particleArray,
		ScalarFieldGrid, spatialGridInfo, scalarGridInfo, params);
}

//! func: compact the surface vertices into a continuous array.(discard those that are not in surface region)
//! So we can deal with this compacted array to get higher performance without conditional branch.
__global__ 
void compactSurfaceVertex(
	SurfaceVerticesIndexArray svIndexArray,	// output, compacted surface vertices' indices array.
	IsSurfaceGrid isSurfaceGridScan,		// input, exclusive prefix sum of isSurfaceGrid.
	IsSurfaceGrid isSurfaceGrid,			// input, whether the corresponding grid point is in surface region or not.
	SimParam params)			
{
	uint threadId = getThreadIdGlobal();
	if (threadId >= isSurfaceGridScan.size || isSurfaceGrid.grid[threadId] != 1)
		return;
	svIndexArray.grid[isSurfaceGridScan.grid[threadId]] = threadId;
}

//! func: call function updateVertexValue() to calculate scalar field value.
//  This time we don't need to deal with non-surface vertices.
__global__ 
void updateScalarGridValuesCompacted(
	SurfaceVerticesIndexArray svIndexArray,		// input, compacted surface vertices' indices array.
	uint numSurfaceVertices,					// input, length of svIndexArray.
	ParticleIndexRangeGrid particleIndexRangeGrid,// input, particles' indices for each cell of spatial grid.	
	ParticleArray particleArray,				// input, particles' position array.
	ScalarFieldGrid ScalarFieldGrid,						// output, scalar field grid.
	GridInfo spatialGridInfo,					// input, spatial hashing grid information.
	GridInfo scalarGridInfo,					// input, scalar field grid information.
	SimParam params)					
{
	uint threadId = getThreadIdGlobal();
	if (threadId >= svIndexArray.size || threadId >= numSurfaceVertices)
		return;
	//updateScalarFieldValueTC01(svIndexArray.grid[threadId], particleIndexRangeGrid, particleArray,
	//	ScalarFieldGrid, spatialGridInfo, scalarGridInfo, params);
	updateScalarFieldValueZB05(svIndexArray.grid[threadId], particleIndexRangeGrid, particleArray,
		ScalarFieldGrid, spatialGridInfo, scalarGridInfo, params);
}

//! func: valid surface cubes detection. Here "valud" means that the cube will produce triangles.
//! We detect those valid cubes to avoid extra branch.
__global__ 
void detectValidSurfaceCubes(
	SurfaceVerticesIndexArray svIndexArray,		// input, compacted surface vertices' indices array.
	uint numSurfaceVertices,					// input, length of svIndexArray.
	ScalarFieldGrid vGrid,							// input, scalar field grid.
	IsValidSurfaceGrid isValidSurfaceGrid,		// output, whether the cell is valid or not.
	NumVerticesGrid numVerticesGrid,			// output, number of vertices per cell.
	IsSurfaceGrid isSfGrid,						// input, whether the corresponding grid point is in surface region or not.
	SimParam params)						
{
	uint threadId = getThreadIdGlobal();
	if (threadId >= svIndexArray.size || threadId >= numSurfaceVertices)
		return;
	// get 3D index and boundary handling.
	uint cubeIndex1D = svIndexArray.grid[threadId];
	uint3 cubeIndex3D = index1DTo3D(cubeIndex1D, vGrid.resolution);
	if (cubeIndex3D.x >= vGrid.resolution.x - 1 ||
		cubeIndex3D.y >= vGrid.resolution.y - 1 ||
		cubeIndex3D.z >= vGrid.resolution.z - 1)
		return;
	// get 8 corners of the cube.
	uint cornerIndex1Ds[8];
	getCornerIndex1Ds(cubeIndex3D, vGrid.resolution, cornerIndex1Ds);

	// get corresponding situation flag.
	uint vertexFlag = getVertexFlag(cornerIndex1Ds, vGrid, params.isoValue);

	uint numVertices = 0;
	// �˸����㶼�Ǳ��涥��Ž������ǻ�, �������Ҫ����������˫�������
	if (isAllSfVertex(cornerIndex1Ds, isSfGrid))
	{  
		numVertices = tex1Dfetch(numVerticesTex, vertexFlag);
	}
	
	isValidSurfaceGrid.grid[threadId] = numVertices > 0 ? 1 : 0;
	numVerticesGrid.grid[threadId] = numVertices;
}

//! func: compact the valid cubes into a continuous array.
__global__ 
void compactValidSurfaceCubes(
	ValidSurfaceIndexArray indexArray,			// output, valid cubes' indices array.
	IsValidSurfaceGrid validSurfaceGridScan,	// input, exculsive prefix sum of validSurafceGrid.
	IsValidSurfaceGrid validSurafceGrid,		// input, whether the cell is valid or not.
	SimParam params
)
{
	uint threadId = getThreadIdGlobal();
	if (threadId >= validSurfaceGridScan.size || validSurafceGrid.grid[threadId] != 1)
		return;
	// save the index of corresponding surface vertex's index.
	indexArray.grid[validSurfaceGridScan.grid[threadId]] = threadId;
}

//! func: generate triangles using marching cube algorithm.
__global__ 
void generateTriangles(
	SurfaceVerticesIndexArray surfaceIndexInGridArray,// input, compacted surface vertices' indice array.
	ValidSurfaceIndexArray validIndexInSurfaceArray,// input, valid cubes' indices array.
	GridInfo scGridInfo,							// input, scalar grid information.
	NumVerticesGrid numVerticesGridScan,			// input, exculsive prefix sum of numVerticesGrid.
	ScalarFieldGrid ScalarFieldGrid,							// input, scalar field grid.
	Float3Grid posGrid,								// output, positions of triangles.
	Float3Grid norGrid,								// output, normals of triangles.
	SimParam params)								
{
	// get corresponding index and boundary handling.
	uint threadId = getThreadIdGlobal();
	if (threadId >= validIndexInSurfaceArray.size)
		return;
	// index of compacted surface vertices' indices array.
	uint surfaceIndex = validIndexInSurfaceArray.grid[threadId];
	// 1D index of sclar field grid.
	uint gridIndex = surfaceIndexInGridArray.grid[surfaceIndex];
	// 3D index of scalar field grid.
	uint3 gridIndex3D = index1DTo3D(gridIndex, ScalarFieldGrid.resolution);

	// get corresponding situation flag.
	uint vertexFlag = getVertexFlag(gridIndex3D, ScalarFieldGrid, params.isoValue);
	// get edge flag.
	uint edgeFlag = tex1Dfetch(edgeTex, vertexFlag);
	// get number of vertices.
	uint numVertices = tex1Dfetch(numVerticesTex, vertexFlag);

	// 8 corners' 3D indices of current cube.
	uint3 cornerIndex3Ds[8];
	// 8 corners' positions of current cube.
	float3 cornerPoss[8];
	// 8 corners' normals of current cube.
	float3 cornerNors[8];
	// 12 edges' intersection positions of current cube.
	float3 intersectPoss[12];
	// 12 edges' intersection normals of current cube.
	float3 intersectNormals[12];

	// get 8 corners' 3D indices.
	getCornerIndex3Ds(gridIndex3D, cornerIndex3Ds);
	// get 8 corners' positions.
	getCornerPositions(cornerIndex3Ds, scGridInfo.minPos, scGridInfo.cellSize, cornerPoss);
	// get 8 corners' normals.
	getCornerNormals(cornerIndex3Ds, ScalarFieldGrid, cornerNors);

	float sign = (params.isoValue < 0.0f) ? (-1.0f) : (1.0f);

	for (int i = 0; i < 12; i++)
	{
		// ���Ϊi�ı����ֵ���ཻ
		if (edgeFlag & (1 << i))
		{
			uint start = tex1Dfetch(vertexIndexesOfEdgeTex, i << 1);
			uint end = tex1Dfetch(vertexIndexesOfEdgeTex, (i << 1) + 1);
			uint startIndex = index3DTo1D(cornerIndex3Ds[start], ScalarFieldGrid.resolution);
			uint endIndex = index3DTo1D(cornerIndex3Ds[end], ScalarFieldGrid.resolution);

			float startValue = ScalarFieldGrid.grid[startIndex].value;
			float endValue = ScalarFieldGrid.grid[endIndex].value;
			float lerpFac = getLerpFac(startValue, endValue, params.isoValue);
			intersectPoss[i] = lerp(cornerPoss[start], cornerPoss[end], lerpFac);
			intersectNormals[i] = sign * normalize(lerp(cornerNors[start], cornerNors[end], lerpFac));
		}
	}
	uint numTri = numVertices / 3;
	for (uint i = 0; i < numTri; i++)
	{
		for (uint j = 0; j < 3; j++)
		{
			int edgeIndex = tex1Dfetch(edgeIndexesOfTriangleTex, vertexFlag * 16 + i * 3 + j);
			uint index = numVerticesGridScan.grid[surfaceIndex] + i * 3 + j;
			posGrid.grid[index] = intersectPoss[edgeIndex];
			norGrid.grid[index] = intersectNormals[edgeIndex];
		}
	}
}

//! -----------------------------------------launch functions for cuda kernel functions----------------------------------

extern "C" 
void launchEstimateSurfaceVertices(
	dim3 gridDim_,
	dim3 blockDim_,
	DensityGrid densityGrid,
	IsSurfaceGrid isSurfaceGrid,
	uint scSearchExt,
	SimParam params)
{
	estimateSurfaceVertices << < gridDim_, blockDim_ >> > (densityGrid, isSurfaceGrid, scSearchExt, params);
	hipDeviceSynchronize();
}

extern "C" 
void launchUpdateScalarGridValues(
	dim3 gridDim_,
	dim3 blockDim_,
	IsSurfaceGrid isSurfaceGrid,
	ParticleIndexRangeGrid particleIndexRangeGrid,
	ParticleArray particleArray,
	ScalarFieldGrid ScalarFieldGrid,
	GridInfo spatialGridInfo,
	GridInfo scalarGridInfo,
	SimParam params)
{
	// not good enough.
	updateScalarGridValuesStd << <gridDim_, blockDim_ >> > (isSurfaceGrid, particleIndexRangeGrid,
		particleArray, ScalarFieldGrid, spatialGridInfo, scalarGridInfo, params);
	hipDeviceSynchronize();
}

extern "C" 
void launchCompactSurfaceVertex(
	dim3 gridDim_,
	dim3 blockDim_,
	SurfaceVerticesIndexArray svIndexArray,
	IsSurfaceGrid isSurfaceGridScan,
	IsSurfaceGrid isSurfaceGrid,
	SimParam params)
{
	compactSurfaceVertex << <gridDim_, blockDim_ >> > (svIndexArray, isSurfaceGridScan, isSurfaceGrid, params);
	hipDeviceSynchronize();
}

extern "C" 
void launchUpdateScalarGridValuesCompacted(
	dim3 gridDim_,
	dim3 blockDim_,
	SurfaceVerticesIndexArray svIndexArray,
	uint numSurfaceVertices, 
	ParticleIndexRangeGrid particleIndexRangeGrid,
	ParticleArray particleArray,
	ScalarFieldGrid ScalarFieldGrid,
	GridInfo spatialGridInfo,
	GridInfo scalarGridInfo,
	SimParam params)
{
	// extra branches are avoided.
	updateScalarGridValuesCompacted << <gridDim_, blockDim_ >> > (svIndexArray, numSurfaceVertices,
		particleIndexRangeGrid, particleArray, ScalarFieldGrid, spatialGridInfo, scalarGridInfo, params);
	hipDeviceSynchronize();
}

extern "C" 
void launchDetectValidSurfaceCubes(
	dim3 gridDim_,
	dim3 blockDim_,
	SurfaceVerticesIndexArray svIndexArray,
	uint numSurfaceVertices,
	ScalarFieldGrid vGrid,
	IsValidSurfaceGrid isValidSurfaceGrid,
	NumVerticesGrid numVerticesGrid,
	IsSurfaceGrid isSfGrid,
	SimParam params)
{
	detectValidSurfaceCubes << <gridDim_, blockDim_ >> > (svIndexArray,
		numSurfaceVertices, vGrid, isValidSurfaceGrid, numVerticesGrid, isSfGrid, params);
	hipDeviceSynchronize();
}

extern "C" 
void launchCompactValidSurfaceCubes(
	dim3 gridDim_, 
	dim3 blockDim_,
	ValidSurfaceIndexArray indexArray,
	IsValidSurfaceGrid validSurfaceGridScan,
	IsValidSurfaceGrid validSurfaceGrid,
	SimParam params)
{
	compactValidSurfaceCubes << <gridDim_, blockDim_ >> > (indexArray,
		validSurfaceGridScan, validSurfaceGrid, params);
	hipDeviceSynchronize();
}

extern "C" 
void launchGenerateTriangles(
	dim3 gridDim_,
	dim3 blockDim_,
	SurfaceVerticesIndexArray surfaceIndexInGridArray,
	ValidSurfaceIndexArray validIndexInSurfaceArray,
	GridInfo scGridInfo,
	NumVerticesGrid numVerticesGrid,
	ScalarFieldGrid ScalarFieldGrid,
	Float3Grid posGrid,
	Float3Grid norGrid,
	SimParam params)
{
	generateTriangles << <gridDim_, blockDim_ >> > (surfaceIndexInGridArray,
		validIndexInSurfaceArray, scGridInfo, numVerticesGrid, ScalarFieldGrid, posGrid, norGrid, params);
	hipDeviceSynchronize();
}

extern "C"
uint launchThrustExclusivePrefixSumScan(uint* output, uint* input, uint numElements)
{
	//! exclusive prefix sum.
	thrust::exclusive_scan(
		thrust::device_ptr<uint>(input),
		thrust::device_ptr<uint>(input + numElements),
		thrust::device_ptr<uint>(output));
	hipDeviceSynchronize();

	uint lastElement = 0;
	uint lastElementScan = 0;
	checkCudaErrors(hipMemcpy((void *)&lastElement, (void *)(input + numElements - 1),
		sizeof(uint), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((void *)&lastElementScan, (void *)(output + numElements - 1),
		sizeof(uint), hipMemcpyDeviceToHost));
	uint sum = lastElement + lastElementScan;
	return sum;
}

//! --------------------------------------Spatial grid establish------------------------------------

//! func: calculation of position's corresponding grid pos.
__device__
uint3 calcGridPosKernel(float3 p, GridInfo spatialGridInfo)
{
	uint3 gridPos;
	gridPos.x = floor((p.x - spatialGridInfo.minPos.x) / spatialGridInfo.cellSize);
	gridPos.y = floor((p.y - spatialGridInfo.minPos.y) / spatialGridInfo.cellSize);
	gridPos.z = floor((p.z - spatialGridInfo.minPos.z) / spatialGridInfo.cellSize);
	return gridPos;
}

//! func: 3D index -> 1D index.
__device__
unsigned int calcGridHashKernel(int3 gridPos, GridInfo spatialGridInfo)
{
	gridPos.x = gridPos.x & (spatialGridInfo.resolution.x - 1);
	gridPos.y = gridPos.y & (spatialGridInfo.resolution.y - 1);
	gridPos.z = gridPos.z & (spatialGridInfo.resolution.z - 1);
	return gridPos.z * spatialGridInfo.resolution.x * spatialGridInfo.resolution.y
		+ gridPos.y * spatialGridInfo.resolution.x + gridPos.x;
}

//! func: calculation of particles' hash value.
__global__
void calcParticlesHashKernel(
	unsigned int *gridParticleHash,		// output, array of particles' hash value.
	ParticleArray particles,			// input, particles array.
	uint numParticles,					// input, number of particles.
	DensityGrid densityGrid,			// output, virtual density grid value.
	GridInfo spatialGridInfo)
{
	unsigned int index = getThreadIdGlobal();
	if (index >= numParticles)
		return;

	float3 curPos = particles.grid[index].pos;
	int3 gridPos = getIndex3D(curPos, spatialGridInfo.minPos, spatialGridInfo.cellSize);
	unsigned int hashValue = index3DTo1D(make_uint3(gridPos.x, gridPos.y, gridPos.z), spatialGridInfo.resolution);
	gridParticleHash[index] = hashValue;
	densityGrid.grid[hashValue] = 1.0f;
}

//! func: find start index and end index for each cell.
__global__
void findCellRangeKernel(
	ParticleIndexRangeGrid particlesIndexRangerArray,	// output, each cells' start index and end index.
	uint numParticles,									// input, number of particles.
	uint *gridParticleHash)								// input, particles' hash value array.
{
	thread_block cta = this_thread_block();
	extern __shared__ unsigned int sharedHash[];
	//unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int index = getThreadIdGlobal();
	unsigned int hashValue;

	if (index < numParticles)
	{
		hashValue = gridParticleHash[index];
		sharedHash[threadIdx.x + 1] = hashValue;

		// first thread in block must load neighbor particle hash
		if (index > 0 && threadIdx.x == 0)
			sharedHash[0] = gridParticleHash[index - 1];
	}

	sync(cta);
	
	if (index < numParticles)
	{
		if (index == 0 || hashValue != sharedHash[threadIdx.x])
		{
			particlesIndexRangerArray.grid[hashValue].start = index;
			if (index > 0)
				particlesIndexRangerArray.grid[sharedHash[threadIdx.x]].end = index;
		}

		if (index == numParticles - 1)
			particlesIndexRangerArray.grid[hashValue].end = index + 1;
	}
}

void launchSpatialGridBuilding(
	ParticleArray particlesArray,
	ScalarFieldGrid densitiesArray,
	uint numParticles,
	ParticleIndexRangeGrid particlesIndexRangerArray,
	DensityGrid flagGrid,
	GridInfo spatialGridInfo)
{
	//! memory allocation for particles' hash value's storage.
	uint *dGridParticleHash;
	hipMalloc((void**)&dGridParticleHash, numParticles * sizeof(unsigned int));

	//! calculation of grid dim and block dim.
	unsigned int numThreads, numBlocks;
	numThreads = 256;
	numBlocks = (numParticles % numThreads != 0) ? (numParticles / numThreads + 1) : (numParticles / numThreads);

	//! step1: computation of hash value of particles.
	calcParticlesHashKernel << <numBlocks, numThreads >> > (
		dGridParticleHash, particlesArray, numParticles, flagGrid, spatialGridInfo);
	getLastCudaError("calcParticlesHashKernel");
	hipDeviceSynchronize();

	//! step2: sort the particle according to their hash value.
	thrust::device_ptr<ParticlePosition> posPtr(particlesArray.grid);
	thrust::device_ptr<ScalarValue> denPtr(densitiesArray.grid);
	thrust::sort_by_key(
		thrust::device_ptr<unsigned int>(dGridParticleHash),
		thrust::device_ptr<unsigned int>(dGridParticleHash + numParticles),
		thrust::make_zip_iterator(thrust::make_tuple(posPtr, denPtr)));
	getLastCudaError("sort_by_key");
	hipDeviceSynchronize();

	//! step3: find start index and end index of each cell.
	// 0xffffffff, need to be attentioned.
	unsigned int memSize = sizeof(unsigned int) * (numThreads + 1);
	hipMemset(particlesIndexRangerArray.grid, 0xffffffff, particlesIndexRangerArray.size * sizeof(IndexRange));
	findCellRangeKernel << < numBlocks, numThreads, memSize >> > (particlesIndexRangerArray, numParticles,
		dGridParticleHash);
	getLastCudaError("findCellRangeKernel");
	hipDeviceSynchronize();

	//! freee memory.
	hipFree(dGridParticleHash);
}
