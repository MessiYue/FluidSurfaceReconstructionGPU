#include "Kernel.cuh"

void initSimParam(SimParam *params)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(simParam), params, sizeof(SimParam)));
	hipDeviceSynchronize();
}

void ThrustExclusiveScan(uint *output, uint *input, uint numElements)
{
	thrust::exclusive_scan(
		thrust::device_ptr<uint>(input),
		thrust::device_ptr<uint>(input + numElements),
		thrust::device_ptr<uint>(output));
}

uint ThrustExclusiveScanWrapper(uint* output, uint* input, uint numElements)
{
	ThrustExclusiveScan(output, input, numElements);
	hipDeviceSynchronize();
	uint lastElement = 0;
	uint lastElementScan = 0;
	checkCudaErrors(hipMemcpy((void *)&lastElement, (void *)(input + numElements - 1), 
		sizeof(uint), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((void *)&lastElementScan, (void *)(output + numElements - 1),
		sizeof(uint), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
	uint sum = lastElement + lastElementScan;
	return sum;
}