#include "Kernel.cuh"

void initSimParam(SimParam *params)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(simParam), params, sizeof(SimParam)));
	hipDeviceSynchronize();
}

uint ThrustExclusiveScanWrapper(uint* output, uint* input, uint numElements)
{
	//! exclusive prefix sum.
	thrust::exclusive_scan(
		thrust::device_ptr<uint>(input),
		thrust::device_ptr<uint>(input + numElements),
		thrust::device_ptr<uint>(output));
	hipDeviceSynchronize();

	uint lastElement = 0;
	uint lastElementScan = 0;
	checkCudaErrors(hipMemcpy((void *)&lastElement, (void *)(input + numElements - 1), 
		sizeof(uint), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((void *)&lastElementScan, (void *)(output + numElements - 1),
		sizeof(uint), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
	uint sum = lastElement + lastElementScan;
	return sum;
}