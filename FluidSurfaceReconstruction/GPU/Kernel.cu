#include "Kernel.cuh"

uint ThrustExclusiveScanWrapper(uint* output, uint* input, uint numElements)
{
	//! exclusive prefix sum.
	thrust::exclusive_scan(
		thrust::device_ptr<uint>(input),
		thrust::device_ptr<uint>(input + numElements),
		thrust::device_ptr<uint>(output));
	hipDeviceSynchronize();

	uint lastElement = 0;
	uint lastElementScan = 0;
	checkCudaErrors(hipMemcpy((void *)&lastElement, (void *)(input + numElements - 1), 
		sizeof(uint), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((void *)&lastElementScan, (void *)(output + numElements - 1),
		sizeof(uint), hipMemcpyDeviceToHost));
	uint sum = lastElement + lastElementScan;
	return sum;
}